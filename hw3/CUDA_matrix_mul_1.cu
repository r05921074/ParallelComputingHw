#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

//#define DEBUG

#define L1 1024
#define L2 1024
#define L3 1024

/* ========== Multiple block, Multiple threads ========== */
/* ========== Can change different matrix length and width ========== */
/* ========== B matrix doen't transposed ========== */
/* ========== fixed block dimension as 32 * 32 ========== */
/* ========== Max array length: 1024 due to MaxThread per side is 1024 ========== */

__global__ void MatMulKernel(float *Ad, float *Bd, float *Cd);
void MatMul(float *A, float *B, float *C);


int main(int argc, char *argv[])
{
    float *A, *B, *C, *AxB;
    int pass = 1;
    A = (float *)calloc(L1 * L2, sizeof(float));
    B = (float *)calloc(L2 * L3, sizeof(float));
    C = (float *)calloc(L1 * L3, sizeof(float));
    AxB = (float *)calloc(L1 * L3, sizeof(float));
  
    /* ========== Assign values to array A and B ========== */  
    for (int i = 0; i < L1; ++i) {
        for (int j = 0; j < L2; ++j) {
            A[i * L2 + j] = rand() % 30;
        }
    }
    for (int i = 0; i < L2; ++i) {
        for (int j = 0; j < L3; ++j) {
            B[i * L3 + j] = rand() % 30;
        }
    }

#ifdef DEBUG
    printf("Matrix A:\n");
    for (int i = 0; i < L1; i++) {
        for (int j = 0; j < L2; j++) {
            printf("%3.0f", A[i * L2 + j]);
        }
        printf("\n");
    }
    printf("Matrix B:\n");
    for (int i = 0; i < L2; i++) {
        for (int j = 0; j < L3; j++) {
            printf("%3.0f", B[i * L3 + j]);
        }
        printf("\n");
    }
#endif
    
    /* ========== Calculate correct answers by CPU ========== */
    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);
    for (int i = 0; i < L1; ++i) {
        for (int j = 0; j < L3; ++j) {
            for (int k = 0; k < L2; ++k) {
                AxB[i * L3 + j] += A[i * L2 + k] * B[k * L3 + j];
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);

#ifdef DEBUG
    printf("Matrix AxB:\n");
    for (int i = 0; i < L1; i++) {
        for (int j = 0; j < L3; j++) {
            printf("%5.0f", AxB[i * L3 + j]);
        }
        printf("\n");
    }
#endif
    
    /* ========== Calculate answers by GPU ========== */
    MatMul((float *)A, (float *)B, (float *)C);

#ifdef DEBUG
    printf("Matrix C:\n");
    for (int i = 0; i < L1; i++) {
        for (int j = 0; j < L3; j++) {
            printf("%5.0f", C[i * L3 + j]);
        }
        printf("\n");
    }
#endif
    
    /* ========== Check if answers correct ========== */
    for (int i = 0; i < L1; ++i) {
        for (int j = 0; j < L3; ++j) {
            if(AxB[i * L3 + j] != C[i * L3 + j]) {
				printf("AxB[%d][%d] = %2.0f   C[%d][%d] = %2.0f\n", i, j, AxB[i * L3 + j], i, j, C[i * L3 + j]);
                pass = 0;
            }
        }
    }
    
    printf("Test %s\n", (pass)?"PASSED":"FAILED");

    free(A);
    free(B);
    free(C);
    free(AxB);
    
    return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *Ad, float *Bd, float *Cd)
{
    // Thread row and column within matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < L1 && col < L3) {
        float Cvalue = 0;

        for (int k = 0; k < L2; k++) {
            float Aval = Ad[row * L2 + k];
            float Bval = Bd[k * L3 + col];
            Cvalue += Aval * Bval;
        }

        Cd[row * L3 + col] = Cvalue;
    }
}

/* ========== Matrix multiplication - Host code ========== */
void MatMul(float *A, float *B, float *C)
{
    size_t size_1 = L1 * L2 * sizeof(float);
    size_t size_2 = L2 * L3 * sizeof(float);
    size_t size_3 = L1 * L3 * sizeof(float);
    float *Ad, *Bd, *Cd;
    
    /* ========== Allocate and Load A, B to device memory ========== */
    hipMalloc((void **)&Ad, size_1);
    hipMemcpy(Ad, A, size_1, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&Bd, size_2);
    hipMemcpy(Bd, B, size_2, hipMemcpyHostToDevice);
    
    /* ========== Allocate C on the device ========== */
    hipMalloc((void **)&Cd, size_3);
    
    /* ========== Setup the execution configuration ========== */
    int GridDim_x = (L3 + 31) / 32, GridDim_y = (L1 + 31) / 32;
    dim3 dimGrid(GridDim_x, GridDim_y);
    dim3 dimBlock(32, 32);

    /* ========== Get start time event ========== */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    /* ========== Invoke kernel ========== */
    MatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd);
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }
    
    /* ========== Get stop time event ========== */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    /* ========== Compute execution time ========== */
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    /* ========== Read C from device memory ========== */
    hipMemcpy(C, Cd, size_3, hipMemcpyDeviceToHost);
    
    /* ========== Free device memory ========== */
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}
