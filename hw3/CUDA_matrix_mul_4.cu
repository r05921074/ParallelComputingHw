#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

//#define B_T
//#define DEBUG

#define L1 383
#define L2 103
#define L3 993
#define TILE_WIDTH  32

/* ========== Multiple block, Multiple threads ========== */
/* ========== Tile multiplication, Shared memory ========== */
/* ========== Can change different matrix length and width (with bug now, only accept multiply of TILE_WIDTH)========== */
/* ========== B matrix doesn't transposed ========== */
/* ========== fixed block dimension as 32 * 32 ========== */
/* ========== Max array length: 1024 due to MaxThread per side is 1024 ========== */

__device__ float GetElement(float *matrix, int row, int col, int width);
__device__ void SetElement(float *matrix, int row, int col, int width, float value);
__device__ float *GetSubMatrix(float *matrix, int blockrow, int blockcol, int width);
__global__ void MatMulKernel(float *Ad, float *Bd, float *Cd);
void MatMul(float *A, float *B, float *C);


int main(int argc, char *argv[])
{
    int pass = 1;

    float *A = (float *)calloc(L1 * L2, sizeof(float));
    float *B = (float *)calloc(L2 * L3, sizeof(float));
    float *C = (float *)calloc(L1 * L3, sizeof(float));
    float *AxB = (float *)calloc(L1 * L3, sizeof(float));
    
    /* ========== Assign values to array A and B ========== */  
    for (int i = 0; i < L1; ++i) {
        for (int j = 0; j < L2; ++j) {
            A[i * L2 + j] = rand() % 30;
        }
    }
    for (int i = 0; i < L2; ++i) {
        for (int j = 0; j < L3; ++j) {
            B[i * L3 + j] = rand() % 30;
        }
    }

#ifdef DEBUG
    printf("Matrix A:\n");
    for (int i = 0; i < L1; i++) {
        for (int j = 0; j < L2; j++) {
            printf("%3.0f", A[i * L2 + j]);
        }
        printf("\n");
    }
    printf("Matrix B:\n");
    for (int i = 0; i < L2; i++) {
        for (int j = 0; j < L3; j++) {
            printf("%3.0f", B[i * L3 + j]);
        }
        printf("\n");
    }
#endif
    
    /* ========== Calculate correct answers by CPU ========== */
    struct timeval starttime, endtime;
    gettimeofday(&starttime, NULL);

#ifdef B_T
    for (int i = 0; i < L2; i++) {
        for (int j = 0; j < L3; j++) {
            //B_t[j * L2 + i] = B[i * L3 + j];
        }
    }
#endif

#ifdef DEBUG
    // printf("Matrix B_t:\n");
    // for (int i = 0; i < L3; i++) {
    //     for (int j = 0; j < L2; j++) {
    //         printf("%5.0f", B_t[i * L2 + j]);
    //     }
    //     printf("\n");
    // }
#endif

    for (int i = 0; i < L1; ++i) {
        for (int j = 0; j < L3; ++j) {
            for (int k = 0; k < L2; ++k) {
                #ifdef B_T
                //AxB[i * L3 + j] += A[i * L2 + k] * B_t[j * L2 + k];
                #endif
                #ifndef B_T
                AxB[i * L3 + j] += A[i * L2 + k] * B[k * L3 + j];
                #endif
            }
        }
    }
    gettimeofday(&endtime, NULL);
    double executime;
    executime = (endtime.tv_sec - starttime.tv_sec) * 1000.0;
    executime += (endtime.tv_usec - starttime.tv_usec) / 1000.0;
    printf("CPU time: %13lf msec\n", executime);

#ifdef DEBUG
    printf("Matrix AxB:\n");
    for (int i = 0; i < L1; i++) {
        for (int j = 0; j < L3; j++) {
            printf("%5.0f", AxB[i * L3 + j]);
        }
        printf("\n");
    }
#endif
    
    /* ========== Calculate answers by GPU ========== */
    MatMul((float *)A, (float *)B, (float *)C);

#ifdef DEBUG
    printf("Matrix C:\n");
    for (int i = 0; i < L1; i++) {
        for (int j = 0; j < L3; j++) {
            printf("%5.0f", C[i * L3 + j]);
        }
        printf("\n");
    }
#endif
    
    /* ========== Check if answers correct ========== */
    for (int i = 0; i < L1; ++i) {
        for (int j = 0; j < L3; ++j) {
            if(AxB[i * L3 + j] != C[i * L3 + j]) {
                //printf("AxB[%d][%d] = %2.0f   C[%d][%d] = %2.0f\n", i, j, AxB[i * L3 + j], i, j, C[i * L3 + j]);
                pass = 0;
            }
        }
    }
    
    printf("Test %s\n", (pass)?"PASSED":"FAILED");

    free(A);
    free(B);
    free(C);
    free(AxB);
    
    return 0;
}

// Get a matrix element
__device__ float GetElement(float *matrix, int row, int col, int width)
{
    return *(matrix + row*width + col);
}

// Set a matrix element
__device__ void SetElement(float *matrix, int row, int col, int width, float value)
{
    *(matrix + row*width + col) = value;
}

// Get the TILE_WIDTHxTILE_WIDTH sub-matrix matsub of matrix that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of matrix
__device__ float *GetSubMatrix(float *matrix, int blockrow, int blockcol, int width)
{
    return (matrix + blockrow*TILE_WIDTH*width + blockcol*TILE_WIDTH);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *Ad, float *Bd, float *Cd)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int row = threadIdx.y;
    int col = threadIdx.x;
    int globalRow = blockRow * blockDim.y + row;
    int globalCol = blockCol * blockDim.x + col;

    int iter = (L2 + TILE_WIDTH - 1) / TILE_WIDTH;
    int residue = L2 % TILE_WIDTH;

    __shared__ float shared_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float shared_B[TILE_WIDTH][TILE_WIDTH];
    
    float Cvalue = 0;
    for (int m = 0; m < iter; ++m) {
        if (globalRow < L1 && (m * TILE_WIDTH + col) < L2) 
            shared_A[row][col] = Ad[globalRow * L2 + (m * TILE_WIDTH + col)];
        else
            shared_A[row][col] = 0;

        if ((m * TILE_WIDTH + row) < L2 && globalCol < L3) 
            shared_B[row][col] = Bd[(m * TILE_WIDTH + row) * L3 + globalCol];
        else
            shared_B[row][col] = 0;

        __syncthreads();
        
        for (int k = 0; k < TILE_WIDTH; ++k) {
            float Aelement = shared_A[row][k];
            float Belement = shared_B[k][col];
            Cvalue += Aelement * Belement;
        }

        __syncthreads();
    }

    if (globalRow < L1 && globalCol < L3)
        Cd[globalRow * L3 + globalCol] = Cvalue;
}

/* ========== Matrix multiplication - Host code ========== */
void MatMul(float *A, float *B, float *C)
{
    size_t size_1 = L1 * L2 * sizeof(float);
    size_t size_2 = L2 * L3 * sizeof(float);
    size_t size_3 = L1 * L3 * sizeof(float);
    float *Ad, *Bd, *Cd;

    /* ========== Allocate and Load A, B to device memory ========== */
    hipMalloc((void **)&Ad, size_1);
    hipMemcpy(Ad, A, size_1, hipMemcpyHostToDevice);
    
    hipMalloc((void **)&Bd, size_2);
    hipMemcpy(Bd, B, size_2, hipMemcpyHostToDevice);
    
    /* ========== Allocate C on the device ========== */
    hipMalloc((void **)&Cd, size_3);

    /* ========== Setup the execution configuration ========== */
    int GridDim_x = (L3 + TILE_WIDTH - 1) / TILE_WIDTH;
    int GridDim_y = (L1 + TILE_WIDTH - 1) / TILE_WIDTH;
    printf("%d, %d\n", GridDim_x, GridDim_y);
    dim3 dimGrid(GridDim_x, GridDim_y);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

    /* ========== Get start time event ========== */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    /* ========== Invoke kernel ========== */
    hipError_t cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }

    MatMulKernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd);
    cuda_err = hipGetLastError();
    if ( hipSuccess != cuda_err ){
        printf("before kernel call: error = %s\n", hipGetErrorString (cuda_err));
        exit(1) ;
    }
    
    /* ========== Get stop time event ========== */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    /* ========== Compute execution time ========== */
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    /* ========== Read C from device memory ========== */
    hipMemcpy(C, Cd, size_3, hipMemcpyDeviceToHost);
    
    /* ========== Free device memory ========== */
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}